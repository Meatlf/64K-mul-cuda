#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include "ModP.h"
#include "kernel.h"

using namespace cuHE;
__global__ void dotMul_kernel (uint64_t* z, uint64_t* x, uint64_t* y ){
	int index = threadIdx.x + blockIdx.x * blockDim.x;

		z[index]=x[index]*y[index];
	
}


void dotMul(uint64_t* z, uint64_t* x, uint64_t* y){
	dotMul_kernel<<<256,256>>>(z,x, y);
}
