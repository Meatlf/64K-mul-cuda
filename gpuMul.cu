#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : gpuMul.cu
 Author      : ttz 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "NTT.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdint.h>
#include <stdlib.h>
#include "ModP.h"
#include "kernel.h"
#include "Base.h"
#include <NTL/ZZ.h>
#include <NTL/ZZX.h>
#include <ctime>
using namespace NTL;
using namespace cuHE;
		
#define len 256
#define len_64K 65536
#define len_64K_half 32768
int main(){
	clock_t t1,t2;
//	t1=clock();
	const ZZ P=to_ZZ(0xffffffff00000001);
	const ZZ root_256=to_ZZ((uint64)14041890976876060974);
	const ZZ root_64K=to_ZZ((uint64)15893793146607301539);

	uint32 *hx_l,*dx_l;
	uint64 *ht_l,*dt_l;
	uint64 *hy_l,*dy_l;

	uint32 *hx_r,*dx_r;
	uint64 *ht_r,*dt_r;
	uint64 *hy_r,*dy_r;
  
	uint64 *hx,*dx;
	uint64 *ht,*dt;
	uint64 *hy,*dy;

	uint64 *h_roots,*d_roots;
	uint64 *h_roots_64K,*d_roots_64K;
  uint64 *h_roots_64K_inverse,*d_roots_64K_inverse;  
	
	dim3 BlockDim(16,16);

  hipStream_t stream[2];
	for(int i=0;i<2;i++)
			hipStreamCreate(&stream[i]);
	t1=clock();	
	hipHostMalloc(&hx_l,len_64K*sizeof(uint32));
	hipHostMalloc(&ht_l,len_64K*sizeof(uint64));
	hipHostMalloc(&hy_l,len_64K*sizeof(uint64));
	
  hipHostMalloc(&hx_r,len_64K*sizeof(uint32));
  hipHostMalloc(&ht_r,len_64K*sizeof(uint64));
  hipHostMalloc(&hy_r,len_64K*sizeof(uint64));	

	hipHostMalloc(&hx,len_64K*sizeof(uint64));
	hipHostMalloc(&ht,len_64K*sizeof(uint64));
	hipHostMalloc(&hy,len_64K*sizeof(uint64));

	hipHostMalloc(&h_roots,len*sizeof(uint64));
  hipHostMalloc(&h_roots_64K,len_64K*sizeof(uint64));
	hipHostMalloc(&h_roots_64K_inverse,len_64K*sizeof(uint64));
	t2=clock();
//	t1=clock();
	hipMalloc(&dx_l,len_64K*sizeof(uint32));
	hipMalloc(&dt_l,len_64K*sizeof(uint64));
	hipMalloc(&dy_l,len_64K*sizeof(uint64));

	 hipMalloc(&dx_r,len_64K*sizeof(uint32));
   hipMalloc(&dt_r,len_64K*sizeof(uint64));
   hipMalloc(&dy_r,len_64K*sizeof(uint64));
	 
	 hipMalloc(&dx,len_64K*sizeof(uint64));
	 hipMalloc(&dt,len_64K*sizeof(uint64));
	 hipMalloc(&dy,len_64K*sizeof(uint64));

 	 hipMalloc(&d_roots,len*sizeof(uint64));
	 hipMalloc(&d_roots_64K,len_64K*sizeof(uint64));
	 hipMalloc(&d_roots_64K_inverse,len_64K*sizeof(uint64));	
//		t2=clock();
  for(int i=0;i<len_64K_half;i++){
		hx_l[i]=2;
	 	ht_l[i]=0;
		hy_l[i]=0;
		
		hx_r[i]=2;
		ht_r[i]=0;
		hy_r[i]=0;
		
	  hx[i]=0;
		ht[i]=0;
		hy[i]=0;
	}

  for(int i=0;i<16;i++){
		for(int k=0;k<16;k++){
				conv(h_roots[16*i+k],PowerMod(root_256,i*k,P));
		//test:		cout<<h_roots[16*i+k]<<endl;
		}
	}

	for(int i=0;i<256;i++){
		for(int k=0;k<256;k++){
				conv(h_roots_64K[256*i+k],PowerMod(root_64K,i*k,P));
		}
	}

	for(int i=0;i<256;i++){
		for(int k=0;k<256;k++){
				conv(h_roots_64K_inverse[256*i+k],PowerMod(root_64K,65535*i*k,P));
		}
	}

  	hipMemcpyAsync(dx_l,hx_l,len_64K*sizeof(uint32),hipMemcpyHostToDevice,stream[0]);
		hipMemcpyAsync(dx_r,hx_r,len_64K*sizeof(uint32),hipMemcpyHostToDevice,stream[1]);

  	hipMemcpyAsync(d_roots,h_roots,len*sizeof(uint64),hipMemcpyHostToDevice,stream[0]);
		hipMemcpyAsync(d_roots,h_roots,len*sizeof(uint64),hipMemcpyHostToDevice,stream[1]);
		hipMemcpyAsync(d_roots_64K,h_roots_64K,len_64K*sizeof(uint64),hipMemcpyHostToDevice,stream[0]);
		hipMemcpyAsync(d_roots_64K,h_roots_64K,len_64K*sizeof(uint64),hipMemcpyHostToDevice,stream[1]);
		//t1=clock();	
		NTT_Kernel_64K_points_1_one<<<16,BlockDim,0,stream[0]>>>(dt_l,dx_l,d_roots);
		NTT_Kernel_64K_points_2_one<<<16,BlockDim,0,stream[0]>>>(dy_l,dt_l,d_roots,d_roots_64K);
		NTT_Kernel_64K_points_1_second<<<16,BlockDim,0,stream[1]>>>(dt_r,dx_r,d_roots);
		NTT_Kernel_64K_points_2_second<<<16,BlockDim,0,stream[1]>>>(dy_r,dt_r,d_roots,d_roots_64K);
  	hipMemcpyAsync(hy_l,dy_l,len_64K*sizeof(uint64),hipMemcpyDeviceToHost,stream[0]);
		hipMemcpyAsync(hy_r,dy_r,len_64K*sizeof(uint64),hipMemcpyDeviceToHost,stream[1]);	

		dotMul(hx,hy_l,hy_r);
		hipMemcpy(d_roots_64K_inverse,h_roots_64K_inverse,len_64K*sizeof(uint64),hipMemcpyHostToDevice);
		hipMemcpy(dx,hx,len_64K*sizeof(uint64),hipMemcpyHostToDevice);
		intt_64K(dy,dt,dx,d_roots,d_roots_64K_inverse);
		hipMemcpy(hy,dy,len_64K*sizeof(uint64),hipMemcpyDeviceToHost);
//	t2=clock();
//	cout<<(double)(t2-t1)/CLOCKS_PER_SEC<<endl;
		uint64_t carry=0;
		t1=clock();
		for(int i=0;i<65536;i++){
			hy[i]+=carry;
			uint64_t current=hy[i];
			carry=current>>32;
			hy[i]=hy[i]&0xFFFFFFFF;
 // 		cout<<hy[i]<<endl;
	  }
		t2=clock();
		cout<<(double)(t2-t1)/CLOCKS_PER_SEC<<endl;

	hipFree(dx_l);
	hipFree(dt_l);
	hipFree(dy_l);

	hipFree(dx_r);
	hipFree(dt_r);
	hipFree(dy_r);
	
  hipFree(dx);
	hipFree(dt);	
	hipFree(dy);

	hipFree(d_roots);
	hipFree(d_roots_64K);
	hipFree(d_roots_64K_inverse);
	
	hipHostFree(hx_l);
	hipHostFree(hx_l);
	hipHostFree(hx_l);

	hipHostFree(hx_r);
	hipHostFree(ht_r);
	hipHostFree(hy_r);
	
	hipHostFree(hx);
	hipHostFree(ht);
	hipHostFree(hy);

	hipHostFree(h_roots);
	hipHostFree(h_roots_64K);
	hipHostFree(h_roots_64K_inverse);
	return 0;
}
